#include "hip/hip_runtime.h"
#include <stdexcept>

#include "augment.h"

using namespace augment;

template <typename T> __device__ void store(T &, float);

template <> __device__ void store<float>(float &out, float val) { out = val; }

template <> __device__ void store<uint8_t>(uint8_t &out, float val) { out = 255 * __saturatef(val); }

template <typename out_t> struct SurfaceTypeMap;

template <> struct SurfaceTypeMap<uint8_t> { using type = uchar4; };

template <> struct SurfaceTypeMap<float> { using type = float4; };

/**
 * @brief Loads RGB input tensor into a 2D layered texture.
 *
 * @tparam in_t     Input tensor scalar type (uint8_t or float)
 * @tparam surf_t   Corresponding CUDA surface type (uchar4 or float4)
 * @param input     Input pointer
 * @param output    Output 2D layered surface
 * @param width     Input width in pixels
 * @param height    Input height in pixels
 */
template <typename in_t, typename surf_t = typename SurfaceTypeMap<in_t>::type>
__global__ void loadInputTensor(const in_t *input, hipSurfaceObject_t output, size_t width, size_t height) {
    const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    unsigned int i = 3 * ((blockIdx.z * height + y) * width + x);

    const surf_t rgb_{input[i], input[i + 1], input[i + 2], 0};

    surf2DLayeredwrite(rgb_, output, x * sizeof(surf_t), y, blockIdx.z);
}

template <typename out_t>
__global__ void process(hipTextureObject_t texObj,
                        out_t *out,
                        const size_t width,
                        const size_t height,
                        const size_t batchSize,
                        const Params *params) {
    // get pixel position
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height)
        return;

    const auto &imgParams = params[blockIdx.z];

    // calculate normalized texture coordinates
    float u = ((float)x + 0.5f) / width;
    float v = ((float)y + 0.5f) / height;

    // apply translation
    u += imgParams.translation[0];
    v += imgParams.translation[1];

    // apply 3D rotation
    u -= 0.5f;
    v -= 0.5f;
    float z = imgParams.geom[2][2] / (imgParams.geom[2][2] + imgParams.geom[0][2] * u + imgParams.geom[1][2] * v);
    // solving for z of intersection of the outbound ray with the image plane
    float tu = (z * (u * imgParams.geom[0][0] + v * imgParams.geom[1][0]) + (z - 1) * imgParams.geom[2][0]) + 0.5f;
    float tv = (z * (u * imgParams.geom[0][1] + v * imgParams.geom[1][1]) + (z - 1) * imgParams.geom[2][1]) + 0.5f;

    // apply flipping
    if (imgParams.flags & FLAG_HORIZONTAL_FLIP)
        tu = 1.0f - tu;
    if (imgParams.flags & FLAG_VERTICAL_FLIP)
        tv = 1.0f - tv;

    // sample the input texture
    float4 sample = tex2DLayered<float4>(texObj, tu, tv, blockIdx.z);

    // get another sample (Mixup)
    if (blockIdx.z != imgParams.mixImgIdx) {
        if (imgParams.flags & FLAG_MIX_HORIZONTAL_FLIP)
            tu = 1.0f - tu;
        if (imgParams.flags & FLAG_MIX_VERTICAL_FLIP)
            tv = 1.0f - tv;

        float4 sample2 = tex2DLayered<float4>(texObj, tu, tv, imgParams.mixImgIdx);

        sample.x = (1 - imgParams.mixFactor) * sample.x + imgParams.mixFactor * sample2.x;
        sample.y = (1 - imgParams.mixFactor) * sample.y + imgParams.mixFactor * sample2.y;
        sample.z = (1 - imgParams.mixFactor) * sample.z + imgParams.mixFactor * sample2.z;
    }

    // fill surroundings
    if (tu <= 0.0f || tu >= 1.0f || tv <= 0.0f || tv >= 1.0f)
        sample.x = sample.y = sample.z = 0.5f;

    // cutout
    if (imgParams.flags & FLAG_CUTOUT) {
        if (abs(tu - imgParams.cutoutPos[0]) < imgParams.cutoutSize[0] &&
            abs(tv - imgParams.cutoutPos[1]) < imgParams.cutoutSize[1])
            sample.x = sample.y = sample.z = 0.5f;
    }

    // apply color transform
    if (imgParams.gammaCorr != 1) {
        sample.x = __powf(sample.x, imgParams.gammaCorr);
        sample.y = __powf(sample.y, imgParams.gammaCorr);
        sample.z = __powf(sample.z, imgParams.gammaCorr);
    }
    float r = imgParams.color[0][0] * sample.x + imgParams.color[0][1] * sample.y + imgParams.color[0][2] * sample.z;
    float g = imgParams.color[1][0] * sample.x + imgParams.color[1][1] * sample.y + imgParams.color[1][2] * sample.z;
    float b = imgParams.color[2][0] * sample.x + imgParams.color[2][1] * sample.y + imgParams.color[2][2] * sample.z;

    if (imgParams.flags & FLAG_COLOR_INVERSION) {
        r = 1 - r;
        g = 1 - g;
        b = 1 - b;
    }

    // write out
    unsigned int i = 3 * ((blockIdx.z * height + y) * width + x);
    store(out[i], r);
    store(out[i + 1], g);
    store(out[i + 2], b);
}

template <typename in_t, typename out_t>
void compute(hipStream_t stream,
             const in_t *input,
             out_t *output,
             size_t inWidth,
             size_t inHeight,
             size_t outWidth,
             size_t outHeight,
             size_t batchSize,
             const Params *params) {

    // allocate temporary cuda array
    hipChannelFormatDesc channelDesc;
    channelDesc.f = std::is_same<in_t, float>::value ? hipChannelFormatKindFloat : hipChannelFormatKindUnsigned;
    channelDesc.w = 8 * sizeof(in_t);
    channelDesc.x = 8 * sizeof(in_t);
    channelDesc.y = 8 * sizeof(in_t);
    channelDesc.z = 8 * sizeof(in_t);
    hipArray *buffer;
    auto error = hipMalloc3DArray(&buffer,
                                   &channelDesc,
                                   make_hipExtent(inWidth, inHeight, batchSize),
                                   hipArrayLayered | hipArraySurfaceLoadStore);
    if (error != hipSuccess)
        throw std::runtime_error("Cannot allocate 3D array: " + std::string(hipGetErrorString(error)));

    // prepare the corresponding resource descriptor
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = buffer;

    // make a surface object for input conversion
    hipSurfaceObject_t surfObj;
    error = hipCreateSurfaceObject(&surfObj, &resDesc);
    if (error != hipSuccess) {
        hipFreeArray(buffer);
        throw std::runtime_error("Cannot create surface object: " + std::string(hipGetErrorString(error)));
    }

    // load input tenso to the temporary buffer array
    {
        const dim3 threads(16, 16, 1);
        const dim3 blocks((inWidth + threads.x - 1) / threads.x, (inHeight + threads.y - 1) / threads.y, batchSize);
        loadInputTensor<in_t><<<blocks, threads, 0, stream>>>(input, surfObj, inWidth, inHeight);
    }

    // set up input texture
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeClamp;
    texDesc.addressMode[1] = hipAddressModeClamp;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = std::is_same<in_t, float>::value ? hipReadModeElementType : hipReadModeNormalizedFloat;
    texDesc.normalizedCoords = 1;

    hipTextureObject_t texObj = 0;
    error = hipCreateTextureObject(&texObj, &resDesc, &texDesc, nullptr);
    if (error != hipSuccess) {
        hipFreeArray(buffer);
        throw std::runtime_error("Cannot create texture object: " + std::string(hipGetErrorString(error)));
    }

    // run the main kernel
    const dim3 threads(16, 16, 1);
    const dim3 blocks((outWidth + threads.x - 1) / threads.x, (outHeight + threads.y - 1) / threads.y, batchSize);
    process<<<blocks, threads, 0, stream>>>(texObj, output, outWidth, outHeight, batchSize, params);

    // destroy texture
    hipDestroyTextureObject(texObj);
    hipDestroyTextureObject(surfObj);
    hipFreeArray(buffer);

    // check for errors
    error = hipGetLastError();
    if (error != hipSuccess)
        throw std::runtime_error(hipGetErrorString(error));
}

template <>
void augment::compute(hipStream_t stream,
                      const uint8_t *input,
                      uint8_t *output,
                      size_t inWidth,
                      size_t inHeight,
                      size_t outWidth,
                      size_t outHeight,
                      size_t batchSize,
                      const Params *params) {
    ::compute(stream, input, output, inWidth, inHeight, outWidth, outHeight, batchSize, params);
}

template <>
void augment::compute(hipStream_t stream,
                      const uint8_t *input,
                      float *output,
                      size_t inWidth,
                      size_t inHeight,
                      size_t outWidth,
                      size_t outHeight,
                      size_t batchSize,
                      const Params *params) {
    ::compute(stream, input, output, inWidth, inHeight, outWidth, outHeight, batchSize, params);
}

template <>
void augment::compute(hipStream_t stream,
                      const float *input,
                      float *output,
                      size_t inWidth,
                      size_t inHeight,
                      size_t outWidth,
                      size_t outHeight,
                      size_t batchSize,
                      const Params *params) {
    ::compute(stream, input, output, inWidth, inHeight, outWidth, outHeight, batchSize, params);
}

void augment::setColorTransform(
    Params &params, float hueShiftRad, float saturationFactor, float valueFactor, const float whiteBalanceScales[2]) {
    // Sampling a rotation and scaling matrix in RGB space:
    //   - rotation around (1,1,1) vector by hueShiftRad radians,
    //   - scaling along (1,1,1) vector by valueFactor and in orthogonal
    //   direction by saturationFactor
    static const float sqrt3 = sqrtf(3);
    const float c = cosf(hueShiftRad), s = sinf(hueShiftRad);
    const float _1 = (valueFactor * (12 * saturationFactor * c + 6)) / 18,
                _2 = (valueFactor * (6 * saturationFactor * c + 6 * sqrt3 * saturationFactor * s - 6)) / 18,
                _3 = (valueFactor * (6 * sqrt3 * saturationFactor * s - 6 * saturationFactor * c + 6)) / 18;

    params.color[0][0] = _1;
    params.color[0][1] = -_2;
    params.color[0][2] = _3;

    params.color[1][0] = _3;
    params.color[1][1] = _1;
    params.color[1][2] = -(valueFactor * (saturationFactor * c + sqrt3 * saturationFactor * s - 1)) / 3;

    params.color[2][0] = -_2;
    params.color[2][1] = _3;
    params.color[2][2] = (valueFactor * (4 * saturationFactor * c + 2)) / 6;

    // pre-multiply by white balance scales
    params.color[0][0] *= whiteBalanceScales[0];
    params.color[1][0] *= whiteBalanceScales[0];
    params.color[2][0] *= whiteBalanceScales[0];

    params.color[0][2] *= whiteBalanceScales[0];
    params.color[1][2] *= whiteBalanceScales[0];
    params.color[2][2] *= whiteBalanceScales[0];
}

void augment::setGeometricTransform(Params &params, float pan, float tilt, float roll, float scaleX, float scaleY) {
    /*
        X, Y: image axes, Z: forward

        Rotation in XZ plane (tilt):
            [ cos(a)  0  sin(a)
                   0  1       0
             -sin(a)  0  cos(a) ]

        Rotation in YZ plane (pan):
            [ 1        0      0
              0   cos(b) sin(b)
              0  -sin(b) cos(b) ]

        Rotation in XY plane (roll):
            [  cos(c) sin(c)  0
              -sin(c) cos(c)  0
                    0      0  1 ]

        Considering the image is on Z=0 plane, the camera is at Z=-1 point

        full transform R = XY * YZ * XZ * Diag(scaleX, scaleY, 1)
    */

    const float cosA = std::cos(pan), sinA = std::sin(pan), cosB = std::cos(tilt), sinB = std::sin(tilt),
                cosC = std::cos(roll), sinC = std::sin(roll);

    params.geom[0][0] = -sinA * sinB * sinC + cosA * cosC;
    params.geom[0][1] = sinC * cosB;
    params.geom[0][2] = sinA * cosC + sinB * sinC * cosA;

    params.geom[1][0] = -sinA * sinB * cosC - sinC * cosA;
    params.geom[1][1] = cosB * cosC;
    params.geom[1][2] = -sinA * sinC + sinB * cosA * cosC;

    params.geom[2][0] = -sinA * cosB;
    params.geom[2][1] = -sinB;
    params.geom[2][2] = cosA * cosB;

    // apply scaling factors
    params.geom[0][0] *= scaleX;
    params.geom[0][1] *= scaleX;
    params.geom[0][2] *= scaleX;

    params.geom[1][0] *= scaleY;
    params.geom[1][1] *= scaleY;
    params.geom[1][2] *= scaleY;
}